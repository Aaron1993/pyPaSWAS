
#include <hip/hip_runtime.h>
/** Direction definitions for the direction matrix. These are needed for the trace back */
#define NO_DIRECTION ${NO_DIRECTION}
#define STOP_DIRECTION ${STOP_DIRECTION}

#define UPPER_LEFT_DIRECTION ${UP_LEFT_DIRECTION}
#define UPPER_DIRECTION ${UP_DIRECTION}
#define LEFT_DIRECTION ${LEFT_DIRECTION}

/** Direction definitions for affine gap penalty **/
#define MAIN_MAIN_DIRECTION ${UP_LEFT_DIRECTION}
#define MAIN_I_DIRECTION ${UP_DIRECTION}
#define MAIN_J_DIRECTION ${LEFT_DIRECTION}
#define MAIN_MASK 3

#define I_MAIN_DIRECTION (${UP_LEFT_DIRECTION} << 2)
#define I_I_DIRECTION (${UP_DIRECTION} <<2)
#define I_J_DIRECTION (${LEFT_DIRECTION} <<2)
#define I_MASK 12

#define J_MAIN_DIRECTION (${UP_LEFT_DIRECTION} << 2)
#define J_I_DIRECTION (${UP_DIRECTION} << 2)
#define J_J_DIRECTION (${LEFT_DIRECTION} <<2)
#define J_MASK 48
