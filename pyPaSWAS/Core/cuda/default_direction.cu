
#include <hip/hip_runtime.h>
/** Direction definitions for the direction matrix. These are needed for the trace back */
#define NO_DIRECTION ${NO_DIRECTION}
#define UPPER_LEFT_DIRECTION ${UP_LEFT_DIRECTION}
#define UPPER_DIRECTION ${UP_DIRECTION}
#define LEFT_DIRECTION ${LEFT_DIRECTION}
#define STOP_DIRECTION ${STOP_DIRECTION}

/** Direction definitions for affine gap penalty **/
#define NO_DIRECTION_I ${NO_DIRECTION}
#define UPPER_LEFT_DIRECTION_I (${UP_LEFT_DIRECTION} << 2)
#define UPPER_DIRECTION_I (${UP_DIRECTION} << 2)
#define LEFT_DIRECTION_I (${LEFT_DIRECTION} << 2)
#define STOP_DIRECTION_I ${STOP_DIRECTION}

#define NO_DIRECTION_J ${NO_DIRECTION}
#define UPPER_LEFT_DIRECTION_J (${UP_LEFT_DIRECTION} << 4)
#define UPPER_DIRECTION_J (${UP_DIRECTION} << 4)
#define LEFT_DIRECTION_J (${LEFT_DIRECTION} << 4)
#define STOP_DIRECTION_J ${STOP_DIRECTION}
