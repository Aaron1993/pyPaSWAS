#include "hip/hip_runtime.h"
#include <math.h>
#include <builtin_types.h>

#define INDEX_SIZE ${size}
#define BLOCK_SIZE ${block}
#define STEP_SIZE ${stepSize}

extern "C"
__global__ void calculateDistance(int *index, int *query, float *distances, unsigned int *validComps,
		unsigned int *seqs,
		unsigned int *indexIncrement,
		float scale, unsigned int numSeqs, unsigned int length, float sliceDistance);

__global__ void calculateDistance(int *index, int *query, float *distances, unsigned int *validComps,
		unsigned int *seqs,
		unsigned int *indexIncrement, float scale, unsigned int numSeqs, unsigned int length, float sliceDistance){

	__shared__ float s_distances[INDEX_SIZE];

	unsigned int seq = blockIdx.y / BLOCK_SIZE;
	unsigned int blockY = blockIdx.y % BLOCK_SIZE;
	//unsigned int indexSeq = seq * STEP_SIZE;
	unsigned int comp = blockIdx.x * BLOCK_SIZE + blockY;
	unsigned int block = (comp)*(INDEX_SIZE+1);
	unsigned int threadPlus1 = threadIdx.x+1;
	unsigned int thread = threadIdx.x;
	if (comp < length) {
		s_distances[thread] = (float) index[block+threadPlus1] - (float)query[threadPlus1+(seq*(INDEX_SIZE+1))];
		s_distances[thread] *= s_distances[thread];


		unsigned int offset = 1;

		for (int i=INDEX_SIZE>>1; i > 0 ; i >>=1 ) {
			__syncthreads();
			if (thread < i) {
				int ai = offset*(2*thread+1)-1;
				int bi = offset*(2*thread+2)-1;
				s_distances[bi] += s_distances[ai];
			}
			offset *= 2;
		}
		if (thread == 0){
			s_distances[INDEX_SIZE-1] = sqrt(s_distances[INDEX_SIZE-1])/scale;
			if (s_distances[INDEX_SIZE-1] <= sliceDistance){
				unsigned int index = atomicAdd(indexIncrement, 1);
				distances[index] =  s_distances[INDEX_SIZE-1];
				validComps[index] = comp;
				seqs[index] = seq;
			}
		}

	}
}
