#include "hip/hip_runtime.h"
#include <math.h>
#include <builtin_types.h>

#define INDEX_SIZE ${size}
#define BLOCK_SIZE ${block}


extern "C"
__global__ void calculateDistance(int *index, int *query, float *distances, float scale);

__global__ void calculateDistance(int *index, int *query, float *distances, float scale){

	__shared__ float s_distances[INDEX_SIZE];

	unsigned int block = (blockIdx.x * BLOCK_SIZE + blockIdx.y)*(INDEX_SIZE+1);
	unsigned int threadPlus1 = threadIdx.x+1;
	unsigned int thread = threadIdx.x;
	s_distances[thread] = (float) index[block+threadPlus1] - (float)query[threadPlus1];
	s_distances[thread] *= s_distances[thread];


	unsigned int offset = 1;

	for (int i=INDEX_SIZE>>1; i > 0 ; i >>=1 ) {
		__syncthreads();
		if (thread < i) {
		    int ai = offset*(2*thread+1)-1;
		    int bi = offset*(2*thread+2)-1;
		    s_distances[bi] += s_distances[ai];
		}
		offset *= 2;
	}
	if (thread == 0){
		distances[blockIdx.x*BLOCK_SIZE+blockIdx.y] =  sqrt(s_distances[INDEX_SIZE-1])/scale;
	}
}
